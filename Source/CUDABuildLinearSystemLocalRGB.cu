#include "hip/hip_runtime.h"
﻿#include <cutil_inline.h>
#include <cutil_math.h>
#include <hip/device_functions.h>

#include "cuda_SimpleMatrixUtil.h"
#include "ICPUtil.h"
#include "CameraTrackingInput.h"

/////////////////////////////////////////////////////
// Defines
/////////////////////////////////////////////////////

#define _DEBUG

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 256
#endif

#ifndef ARRAY_SIZE
#define ARRAY_SIZE 30
#endif

#define MINF __int_as_float(0xff800000)

/////////////////////////////////////////////////////
// Shared Memory
/////////////////////////////////////////////////////

__shared__ float bucket4[ARRAY_SIZE*BLOCK_SIZE];

/////////////////////////////////////////////////////
// Helper Functions
/////////////////////////////////////////////////////

__device__ inline void addToLocalScanElement(uint inpGTid, uint resGTid, volatile float* shared)
{
#pragma unroll
	for (uint i = 0; i < ARRAY_SIZE; i++)
	{
		shared[ARRAY_SIZE*resGTid + i] += shared[ARRAY_SIZE*inpGTid + i];
	}
}

__device__ inline void CopyToResultScanElement(uint GID, float* output)
{
#pragma unroll
	for (uint i = 0; i < ARRAY_SIZE; i++)
	{
		output[ARRAY_SIZE*GID + i] = bucket4[0 + i];
	}
}

__device__ inline void CopyToResultScanElementGlobal(uint GID, float* output, float *d_gtemp)
{
#pragma unroll
	for (uint i = 0; i < ARRAY_SIZE; i++)
	{
		output[ARRAY_SIZE*GID + i] = d_gtemp[GID * ARRAY_SIZE * BLOCK_SIZE + i];
	}
}

__device__ inline void CopyToResultScanElementGlobal(uint GID, float* output, float *d_gtemp, float lambda, float3 &stepRot, float3 &stepTrans)
{

#pragma unroll
	for (uint i = 0; i < ARRAY_SIZE; i++)
	{
		output[ARRAY_SIZE*GID + i] = d_gtemp[GID * ARRAY_SIZE * BLOCK_SIZE + i];
	}

	output[ARRAY_SIZE*GID + 0] += lambda;//6
	output[ARRAY_SIZE*GID + 6] += lambda;//5  = 11
	output[ARRAY_SIZE*GID + 11] += lambda;//4 = 15
	output[ARRAY_SIZE*GID + 15] += lambda;//3 = 18
	output[ARRAY_SIZE*GID + 18] += lambda;//2 = 20
	output[ARRAY_SIZE*GID + 20] += lambda;//1
	output[ARRAY_SIZE*GID + 21] -= lambda * stepRot.x;
	output[ARRAY_SIZE*GID + 22] -= lambda * stepRot.y;
	output[ARRAY_SIZE*GID + 23] -= lambda * stepRot.z;
	output[ARRAY_SIZE*GID + 24] -= lambda * stepTrans.x;
	output[ARRAY_SIZE*GID + 25] -= lambda * stepTrans.y;
	output[ARRAY_SIZE*GID + 26] -= lambda * stepTrans.z;
	output[ARRAY_SIZE*GID + 27] += lambda * (dot(stepRot, stepRot) + dot(stepTrans, stepTrans));
	output[ARRAY_SIZE*GID + 28] += lambda;
	output[ARRAY_SIZE*GID + 29] += 1;

}


__device__ inline void addToLocalScanElementGlobal(uint nodeInd, uint inpGTid, uint resGTid, float* d_gtemp)
{
#pragma unroll
	for (uint i = 0; i < ARRAY_SIZE; i++)
	{
		d_gtemp[nodeInd* ARRAY_SIZE * BLOCK_SIZE + ARRAY_SIZE * resGTid + i] += d_gtemp[nodeInd* ARRAY_SIZE * BLOCK_SIZE + ARRAY_SIZE * inpGTid + i];
	}
}


__device__ inline void SetZeroScanElement(uint GTid)
{
#pragma unroll
	for (uint i = 0; i < ARRAY_SIZE; i++)
	{
		bucket4[GTid*ARRAY_SIZE + i] = 0.0f;
	}
}

__device__ inline float weightDist(float2 src, float2 tar, float sigma) {
	float2 diff = src - tar;

	return exp(-(diff.x * diff.x + diff.y * diff.y) / (2 * sigma*sigma));

}

/////////////////////////////////////////////////////
// Scan
/////////////////////////////////////////////////////

__device__ inline void warpReduce(int GTid) // See Optimizing Parallel Reduction in CUDA by Mark Harris
{
	addToLocalScanElement(GTid + 32, GTid, bucket4);
	addToLocalScanElement(GTid + 16, GTid, bucket4);
	addToLocalScanElement(GTid + 8, GTid, bucket4);
	addToLocalScanElement(GTid + 4, GTid, bucket4);
	addToLocalScanElement(GTid + 2, GTid, bucket4);
	addToLocalScanElement(GTid + 1, GTid, bucket4);
}

__device__ inline void warpReduceGlobal(int nodeInd, int GTid, float* d_gtemp) // See Optimizing Parallel Reduction in CUDA by Mark Harris
{
	addToLocalScanElementGlobal(nodeInd, GTid + 32, GTid, d_gtemp);
	//	__syncthread();
	addToLocalScanElementGlobal(nodeInd, GTid + 16, GTid, d_gtemp);
	addToLocalScanElementGlobal(nodeInd, GTid + 8, GTid, d_gtemp);
	addToLocalScanElementGlobal(nodeInd, GTid + 4, GTid, d_gtemp);
	addToLocalScanElementGlobal(nodeInd, GTid + 2, GTid, d_gtemp);
	addToLocalScanElementGlobal(nodeInd, GTid + 1, GTid, d_gtemp);
}

/////////////////////////////////////////////////////
// Compute Normal Equations
/////////////////////////////////////////////////////

__device__ inline  void addToLocalSystemGlobal(mat1x6& jacobianBlockRow, mat1x1& residualsBlockRow, float weight, uint blockInd, uint threadIdx, float* d_gtemp)
{
	uint linRowStart = 0;

#pragma unroll
	for (uint i = 0; i < 6; i++)
	{

		mat1x1 colI; jacobianBlockRow.getBlock(0, i, colI);

#pragma unroll
		for (uint j = i; j < 6; j++)
		{
			mat1x1 colJ; jacobianBlockRow.getBlock(0, j, colJ);

			d_gtemp[blockInd * ARRAY_SIZE* BLOCK_SIZE + ARRAY_SIZE * threadIdx + linRowStart + j - i] += colI.getTranspose()*colJ*weight;
		}

		linRowStart += 6 - i;

		d_gtemp[blockInd * ARRAY_SIZE* BLOCK_SIZE + ARRAY_SIZE * threadIdx + 21 + i] -= colI.getTranspose()*residualsBlockRow*weight; // -JTF

	}

	d_gtemp[blockInd * ARRAY_SIZE* BLOCK_SIZE + ARRAY_SIZE * threadIdx + 27] += weight * residualsBlockRow.norm1DSquared(); // residual
	d_gtemp[blockInd * ARRAY_SIZE* BLOCK_SIZE + ARRAY_SIZE * threadIdx + 28] += weight;									 // weight
	d_gtemp[blockInd * ARRAY_SIZE* BLOCK_SIZE + ARRAY_SIZE * threadIdx + 29] += 1.0f;									 // corr number
}

__device__ inline  void addToLocalSystemGlobal3(mat3x6& jacobianBlockRow, mat3x1& residualsBlockRow, float weight, uint blockInd, uint threadIdx, float* d_gtemp)
{
	uint linRowStart = 0;

#pragma unroll
	for (uint i = 0; i < 6; i++)
	{

		mat3x1 colI; jacobianBlockRow.getBlock(0, i, colI);

#pragma unroll
		for (uint j = i; j < 6; j++)
		{
			mat3x1 colJ; jacobianBlockRow.getBlock(0, j, colJ);

			d_gtemp[blockInd * ARRAY_SIZE* BLOCK_SIZE + ARRAY_SIZE * threadIdx + linRowStart + j - i] += colI.getTranspose()*colJ*weight;
		}

		linRowStart += 6 - i;

		d_gtemp[blockInd * ARRAY_SIZE* BLOCK_SIZE + ARRAY_SIZE * threadIdx + 21 + i] -= colI.getTranspose()*residualsBlockRow*weight; // -JTF

	}

	d_gtemp[blockInd * ARRAY_SIZE* BLOCK_SIZE + ARRAY_SIZE * threadIdx + 27] += weight * residualsBlockRow.norm1DSquared(); // residual
	d_gtemp[blockInd * ARRAY_SIZE* BLOCK_SIZE + ARRAY_SIZE * threadIdx + 28] += weight;									 // weight
	d_gtemp[blockInd * ARRAY_SIZE* BLOCK_SIZE + ARRAY_SIZE * threadIdx + 29] += 1.0f;									 // corr number
}

__device__ inline  void addToLocalSystemGlobal4(matNxM<4, 6>& jacobianBlockRow, mat3x1& residualsNormBlockRow, float residualsBlockColor, float weightColor, float weightNormal, uint blockInd, uint threadIdx, float* d_gtemp)
{
	uint linRowStart = 0;

#pragma unroll
	for (uint i = 0; i < 6; i++)
	{

		mat4x1 colI; jacobianBlockRow.getBlock(0, i, colI);
		mat3x1 colINorm; colI.getBlock(0, 0, colINorm);
#pragma unroll
		for (uint j = i; j < 6; j++)
		{
			mat4x1 colJ; jacobianBlockRow.getBlock(0, j, colJ);
			mat3x1 colJNorm; colJ.getBlock(0, 0, colJNorm);

			d_gtemp[blockInd * ARRAY_SIZE* BLOCK_SIZE + ARRAY_SIZE * threadIdx + linRowStart + j - i] += (((colINorm.getTranspose()*colJNorm)*weightNormal)(0) + (colI(3) * colJ(3) * weightColor));
		}

		linRowStart += 6 - i;

		d_gtemp[blockInd * ARRAY_SIZE* BLOCK_SIZE + ARRAY_SIZE * threadIdx + 21 + i] -= (((colINorm.getTranspose()*residualsNormBlockRow)*weightNormal)(0) + (colI(3) * residualsBlockColor) * weightColor); // -JTF

	}

	d_gtemp[blockInd * ARRAY_SIZE* BLOCK_SIZE + ARRAY_SIZE * threadIdx + 27] += (weightNormal * residualsNormBlockRow.norm1DSquared() + residualsBlockColor * residualsBlockColor * weightColor); // residual
	d_gtemp[blockInd * ARRAY_SIZE* BLOCK_SIZE + ARRAY_SIZE * threadIdx + 28] += (weightNormal + weightColor);									 // weight
	d_gtemp[blockInd * ARRAY_SIZE* BLOCK_SIZE + ARRAY_SIZE * threadIdx + 29] += 1.0f;									 // corr number
}

__device__ inline  void addToLocalSystem(mat1x6& jacobianBlockRow, mat1x1& residualsBlockRow, float weight, uint threadIdx, volatile float* shared)
{
	uint linRowStart = 0;

#pragma unroll
	for (uint i = 0; i < 6; i++)
	{
		mat1x1 colI; jacobianBlockRow.getBlock(0, i, colI);

#pragma unroll
		for (uint j = i; j < 6; j++)
		{
			mat1x1 colJ; jacobianBlockRow.getBlock(0, j, colJ);

			shared[ARRAY_SIZE*threadIdx + linRowStart + j - i] += colI.getTranspose()*colJ*weight;
		}

		linRowStart += 6 - i;

		shared[ARRAY_SIZE*threadIdx + 21 + i] -= colI.getTranspose()*residualsBlockRow*weight; // -JTF
	}

	shared[ARRAY_SIZE*threadIdx + 27] += weight * residualsBlockRow.norm1DSquared(); // residual
	shared[ARRAY_SIZE*threadIdx + 28] += weight;									 // weight

	shared[ARRAY_SIZE*threadIdx + 29] += 1.0f;									 // corr number
}


__global__ void scanNormalEquationsLocalDevice(unsigned int imageWidth, unsigned int imageHeight, float* output, CameraTrackingLocalInput cameraTrackingInput, float3x3 intrinsics, CameraTrackingLocalParameters cameraTrackingIParameters, float3 anglesOld, float3 translationOld, float2 pos, unsigned int localWindowSize)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int localWindowWidth = 2 * cameraTrackingIParameters.localWindowHWidth + 1;
	int2 posi = make_int2(pos.x - cameraTrackingIParameters.localWindowHWidth, pos.y - cameraTrackingIParameters.localWindowHWidth);

	// Set system to zero
	SetZeroScanElement(threadIdx.x);

	//Locally sum small window
#pragma unroll
	for (uint i = 0; i < localWindowSize; i++)
	{
		const int localIndex1D = localWindowSize * x + i;
		int2 index = make_int2(localIndex1D%localWindowWidth, localIndex1D / localWindowWidth);

		index += posi;
		const int index1D = index.x + index.y * imageWidth;


		if (0 <= index.x && index.x < imageWidth && 0 <= index.y && index.y < imageHeight)
		{
			mat3x1 pInput = mat3x1(make_float3(cameraTrackingInput.d_inputPos[index1D]));
			mat3x1 nInput = mat3x1(make_float3(cameraTrackingInput.d_inputNormal[index1D]));
			mat1x1 iInput = mat1x1(cameraTrackingInput.d_inputIntensity[index1D]);


			if (!pInput.checkMINF() && !nInput.checkMINF() && !iInput.checkMINF() && iInput > 1.f / 255.f)
			{
				mat3x3 I = mat3x3(intrinsics);
				mat3x3 ROld = mat3x3(evalRMat(anglesOld));
				mat3x1 pInputTransformed = ROld * pInput + mat3x1(translationOld);
				mat3x1 nInputTransformed = ROld * nInput;

				mat3x3 Ralpha = mat3x3(evalR_dGamma(anglesOld));
				mat3x3 Rbeta = mat3x3(evalR_dBeta(anglesOld));
				mat3x3 Rgamma = mat3x3(evalR_dAlpha(anglesOld));

				mat3x1 pProjTrans = I * pInputTransformed;

				if (pProjTrans(2) > 0.0f)
				{
					mat2x1 uvModel = mat2x1(dehomogenize(pProjTrans));

					mat3x1 iTargetAndDerivative = mat3x1(make_float3(bilinearInterpolationFloat4(uvModel(0), uvModel(1), cameraTrackingInput.d_targetIntensityAndDerivatives, imageWidth, imageHeight)));
					mat1x1 iTarget = mat1x1(iTargetAndDerivative(0)); mat1x1 iTargetDerivUIntensity = mat1x1(iTargetAndDerivative(1)); mat1x1 iTargetDerivVIntensity = mat1x1(iTargetAndDerivative(2));

					mat2x3 PI = dehomogenizeDerivative(pProjTrans);
					mat3x1 phiAlpha = Ralpha * pInputTransformed; mat3x1 phiBeta = Rbeta * pInputTransformed;	mat3x1 phiGamma = Rgamma * pInputTransformed;

					if (!iTarget.checkMINF() && !iTargetDerivUIntensity.checkMINF() && !iTargetDerivVIntensity.checkMINF())
					{

						// Color
						mat1x1 diffIntensity(iTarget - iInput);
						mat1x2 DIntensity; DIntensity(0, 0) = iTargetDerivUIntensity(0); DIntensity(0, 1) = iTargetDerivVIntensity(0);
						if (DIntensity.norm1D() > cameraTrackingIParameters.colorGradiantMin)
						{
							float weightColor = max(0.0f, 1.0f - diffIntensity.norm1D() / cameraTrackingIParameters.colorThres);

							weightColor = weightDist(make_float2(index.x, index.y), pos, cameraTrackingIParameters.sigma);

							mat1x3 tmp0Intensity = DIntensity * PI*I;

							mat1x6 jacobianBlockRowIntensity;
							jacobianBlockRowIntensity.setBlock(tmp0Intensity*phiAlpha, 0, 0);
							jacobianBlockRowIntensity.setBlock(tmp0Intensity*phiBeta, 0, 1);
							jacobianBlockRowIntensity.setBlock(tmp0Intensity*phiGamma, 0, 2);
							jacobianBlockRowIntensity.setBlock(tmp0Intensity, 0, 3);
							addToLocalSystem(jacobianBlockRowIntensity, diffIntensity, weightColor, threadIdx.x, bucket4);
						}
					}
				}
			}
		}
	}

	__syncthreads();

	// Up sweep 2D
#pragma unroll
	for (unsigned int stride = BLOCK_SIZE / 2; stride > 32; stride >>= 1)
	{
		if (threadIdx.x < stride) addToLocalScanElement(threadIdx.x + stride / 2, threadIdx.x, bucket4);

		__syncthreads();
	}

	if (threadIdx.x < 32) warpReduce(threadIdx.x);

	// Copy to output texture
	if (threadIdx.x == 0) CopyToResultScanElement(blockIdx.x, output);
}

__global__ void scanNormalEquationLocalDevice() {

}

__global__ void scanNormalEquationsLocalParallelDevice(unsigned int imageWidth, unsigned int imageHeight, float* d_system, float *d_temp, float3* d_x_rot, float3* d_x_trans, CameraTrackingLocalInput cameraTrackingInput, float3x3 intrinsics, CameraTrackingLocalParameters cameraTrackingParameters)
{

	unsigned int nodeInd = blockIdx.x;
	unsigned int threadInd = threadIdx.x;

	const int nodeW = cameraTrackingParameters.nodeWidth;
	const int nodeH = cameraTrackingParameters.nodeHeight;
	const int nodeIndX = nodeInd % nodeW;
	const int nodeIndY = nodeInd / nodeW;
	const int imageW = cameraTrackingParameters.imageWidth;
	const int imageH = cameraTrackingParameters.imageHeight;
	const int localWindowHWidth = cameraTrackingParameters.localWindowHWidth;
	const float2 cellWH = cameraTrackingParameters.cellWH;

	float2 nodePos = cellWH * make_float2(nodeIndX, nodeIndY) + cameraTrackingParameters.offset;
	int2 nodePosi = make_int2(nodePos);

	int six = max(0, nodePosi.x - localWindowHWidth);
	int eix = min(imageW - 1, nodePosi.x + localWindowHWidth + 1);
	int siy = max(0, nodePosi.y - localWindowHWidth);
	int eiy = min(imageH - 1, nodePosi.y + localWindowHWidth + 1);

	int rangeW = eix - six; // sx <= x < ex
	int rangeH = eiy - siy; // sy <= y < ey
	int rangeN = rangeW * rangeH;

	for (int i = threadInd; i < rangeN; i += BLOCK_SIZE) {

		int offix = i % rangeW;
		int offiy = i / rangeW;
		int ix = six + offix;
		int iy = siy + offiy;
		int index1D = iy * imageW + ix;

		mat3x1 pInput = mat3x1(make_float3(cameraTrackingInput.d_inputPos[index1D]));
		mat3x1 nInput = mat3x1(make_float3(cameraTrackingInput.d_inputNormal[index1D]));
		mat1x1 iInput = mat1x1(cameraTrackingInput.d_inputIntensity[index1D]);
		float maskVal = cameraTrackingInput.d_inputMask[index1D];

		if (!pInput.checkMINF() && !nInput.checkMINF() && !iInput.checkMINF() && maskVal < 0.5 && iInput > 1.f / 255.f)
		{
			//
			float3 xRot = d_x_rot[nodeInd];
			float3 xTrans = d_x_trans[nodeInd];

			float3 aRot, aTrans;

			mat3x3 I = mat3x3(intrinsics);
			mat3x3 ROld = mat3x3(evalRMat(xRot));
			mat3x1 pInputTransformed = ROld * pInput + mat3x1(xTrans);
			mat3x1 nInputTransformed = ROld * nInput;

			mat3x3 Ralpha = mat3x3(evalR_dGamma(xRot));
			mat3x3 Rbeta = mat3x3(evalR_dBeta(xRot));
			mat3x3 Rgamma = mat3x3(evalR_dAlpha(xRot));

			mat3x1 pProjTrans = I * pInputTransformed;

			if (pProjTrans(2) > 0.0f)
			{
				mat2x1 uvModel = mat2x1(dehomogenize(pProjTrans));

				mat3x1 iTargetAndDerivative = mat3x1(make_float3(bilinearInterpolationFloat4(uvModel(0), uvModel(1), cameraTrackingInput.d_targetIntensityAndDerivatives, imageW, imageH)));
				mat1x1 iTarget = mat1x1(iTargetAndDerivative(0)); mat1x1 iTargetDerivUIntensity = mat1x1(iTargetAndDerivative(1)); mat1x1 iTargetDerivVIntensity = mat1x1(iTargetAndDerivative(2));

				mat2x3 PI = dehomogenizeDerivative(pProjTrans);
				mat3x1 phiAlpha = Ralpha * pInputTransformed; mat3x1 phiBeta = Rbeta * pInputTransformed;	mat3x1 phiGamma = Rgamma * pInputTransformed;

				if (!iTarget.checkMINF() && !iTargetDerivUIntensity.checkMINF() && !iTargetDerivVIntensity.checkMINF())
				{

					// Color
					mat1x1 diffIntensity(iTarget - iInput);
					mat1x2 DIntensity; DIntensity(0, 0) = iTargetDerivUIntensity(0); DIntensity(0, 1) = iTargetDerivVIntensity(0);


					if (DIntensity.norm1D() > cameraTrackingParameters.colorGradiantMin)
					{
						float weightColor;
						//= max(0.0f, 1.0f - diffIntensity.norm1D() / cameraTrackingParameters.colorThres);

						weightColor = weightDist(make_float2(ix, iy), nodePos, cameraTrackingParameters.sigma);
						mat1x3 tmp0Intensity = DIntensity * PI*I;

						mat1x6 jacobianBlockRowIntensity;
						jacobianBlockRowIntensity.setBlock(tmp0Intensity*phiAlpha, 0, 0);
						jacobianBlockRowIntensity.setBlock(tmp0Intensity*phiBeta, 0, 1);
						jacobianBlockRowIntensity.setBlock(tmp0Intensity*phiGamma, 0, 2);
						jacobianBlockRowIntensity.setBlock(tmp0Intensity, 0, 3);
						addToLocalSystemGlobal(jacobianBlockRowIntensity, diffIntensity, weightColor, nodeInd, threadIdx.x, d_temp);

					}
				}
			}
		}
	}

	__syncthreads();


	// Up sweep 2D
#pragma unroll
	for (unsigned int stride = BLOCK_SIZE / 2; stride > 32; stride >>= 1)
	{
		if (threadIdx.x < stride) addToLocalScanElementGlobal(nodeInd, threadIdx.x + stride / 2, threadIdx.x, d_temp);

		__syncthreads();

	}
	if (threadIdx.x < 32) warpReduceGlobal(nodeInd, threadIdx.x, d_temp);


	//
	__syncthreads();

	// Copy to output texture
	if (threadIdx.x == 0) CopyToResultScanElementGlobal(nodeInd, d_system, d_temp);
}

__global__ void scanNormalEquationsLocalParallelRegDevice(unsigned int imageWidth, unsigned int imageHeight, float* d_system, float *d_temp, float3* d_x_rot, float3* d_x_trans, float3* d_x_step_rot, float3* d_x_step_trans, float lambda, CameraTrackingLocalInput cameraTrackingInput, float3x3 intrinsics, CameraTrackingLocalParameters cameraTrackingParameters)
{

	unsigned int nodeInd = blockIdx.x;
	unsigned int threadInd = threadIdx.x;

	const int nodeW = cameraTrackingParameters.nodeWidth;
	const int nodeH = cameraTrackingParameters.nodeHeight;
	const int nodeIndX = nodeInd % nodeW;
	const int nodeIndY = nodeInd / nodeW;
	const int imageW = cameraTrackingParameters.imageWidth;
	const int imageH = cameraTrackingParameters.imageHeight;
	const int localWindowHWidth = cameraTrackingParameters.localWindowHWidth;
	const float2 cellWH = cameraTrackingParameters.cellWH;

	float2 nodePos = cellWH * make_float2(nodeIndX, nodeIndY) + cameraTrackingParameters.offset;
	int2 nodePosi = make_int2(nodePos);

	int six = max(0, nodePosi.x - localWindowHWidth);
	int eix = min(imageW - 1, nodePosi.x + localWindowHWidth + 1);
	int siy = max(0, nodePosi.y - localWindowHWidth);
	int eiy = min(imageH - 1, nodePosi.y + localWindowHWidth + 1);

	int rangeW = eix - six; // sx <= x < ex
	int rangeH = eiy - siy; // sy <= y < ey
	int rangeN = rangeW * rangeH;

	float3 xRot = d_x_rot[nodeInd] + d_x_step_rot[nodeInd];
	float3 xTrans = d_x_trans[nodeInd] + d_x_step_trans[nodeInd];

	for (int i = threadInd; i < rangeN; i += BLOCK_SIZE) {

		int offix = i % rangeW;
		int offiy = i / rangeW;
		int ix = six + offix;
		int iy = siy + offiy;
		int index1D = iy * imageW + ix;

		mat3x1 pInput = mat3x1(make_float3(cameraTrackingInput.d_inputPos[index1D]));
		mat3x1 nInput = mat3x1(make_float3(cameraTrackingInput.d_inputNormal[index1D]));
		mat1x1 iInput = mat1x1(cameraTrackingInput.d_inputIntensity[index1D]);
		float maskVal = cameraTrackingInput.d_inputMask[index1D];

		if (!pInput.checkMINF() && !nInput.checkMINF() && !iInput.checkMINF() && maskVal < 0.5 && iInput > 1.f / 255.f)
		{
			//


			float3 aRot, aTrans;

			mat3x3 I = mat3x3(intrinsics);
			mat3x3 ROld = mat3x3(evalRMat(xRot));
			mat3x1 pInputTransformed = ROld * pInput + mat3x1(xTrans);
			mat3x1 nInputTransformed = ROld * nInput;

			mat3x3 Ralpha = mat3x3(evalR_dGamma(xRot));
			mat3x3 Rbeta = mat3x3(evalR_dBeta(xRot));
			mat3x3 Rgamma = mat3x3(evalR_dAlpha(xRot));

			mat3x1 pProjTrans = I * pInputTransformed;

			if (pProjTrans(2) > 0.0f)
			{
				mat2x1 uvModel = mat2x1(dehomogenize(pProjTrans));

				mat3x1 iTargetAndDerivative = mat3x1(make_float3(bilinearInterpolationFloat4(uvModel(0), uvModel(1), cameraTrackingInput.d_targetIntensityAndDerivatives, imageW, imageH)));
				mat1x1 iTarget = mat1x1(iTargetAndDerivative(0)); mat1x1 iTargetDerivUIntensity = mat1x1(iTargetAndDerivative(1)); mat1x1 iTargetDerivVIntensity = mat1x1(iTargetAndDerivative(2));

				mat2x3 PI = dehomogenizeDerivative(pProjTrans);
				mat3x1 phiAlpha = Ralpha * pInputTransformed; mat3x1 phiBeta = Rbeta * pInputTransformed;	mat3x1 phiGamma = Rgamma * pInputTransformed;

				if (!iTarget.checkMINF() && !iTargetDerivUIntensity.checkMINF() && !iTargetDerivVIntensity.checkMINF())
				{

					// Color
					mat1x1 diffIntensity(iTarget - iInput);
					mat1x2 DIntensity; DIntensity(0, 0) = iTargetDerivUIntensity(0); DIntensity(0, 1) = iTargetDerivVIntensity(0);


					if (DIntensity.norm1D() > cameraTrackingParameters.colorGradiantMin)
					{
						float weightColor;
						weightColor = weightDist(make_float2(ix, iy), nodePos, cameraTrackingParameters.sigma);
						//*max(0.0f, 1.0f - diffIntensity.norm1D() / cameraTrackingParameters.colorThres);;
						mat1x3 tmp0Intensity = DIntensity * PI*I;

						mat1x6 jacobianBlockRowIntensity;
						jacobianBlockRowIntensity.setBlock(tmp0Intensity*phiAlpha, 0, 0);
						jacobianBlockRowIntensity.setBlock(tmp0Intensity*phiBeta, 0, 1);
						jacobianBlockRowIntensity.setBlock(tmp0Intensity*phiGamma, 0, 2);
						jacobianBlockRowIntensity.setBlock(tmp0Intensity, 0, 3);
						addToLocalSystemGlobal(jacobianBlockRowIntensity, diffIntensity, weightColor, nodeInd, threadIdx.x, d_temp);

					}
				}
			}
		}
	}

	__syncthreads();


	// Up sweep 2D
#pragma unroll
	for (unsigned int stride = BLOCK_SIZE / 2; stride > 32; stride >>= 1)
	{
		if (threadIdx.x < stride) addToLocalScanElementGlobal(nodeInd, threadIdx.x + stride / 2, threadIdx.x, d_temp);

		__syncthreads();

	}
	if (threadIdx.x < 32) warpReduceGlobal(nodeInd, threadIdx.x, d_temp);


	//
	__syncthreads();

	// Copy to output texture
	if (threadIdx.x == 0) {

		CopyToResultScanElementGlobal(nodeInd, d_system, d_temp, lambda, d_x_step_rot[nodeInd], d_x_step_trans[nodeInd]);

	}
}

__global__ void scanNormal3EquationsLocalParallelRegDevice(unsigned int imageWidth, unsigned int imageHeight, float* d_system, float *d_temp, float3* d_x_rot, float3* d_x_trans, float3* d_x_step_rot, float3* d_x_step_trans, float lambda, CameraTrackingLocalInput cameraTrackingInput, float3x3 intrinsics, CameraTrackingLocalParameters cameraTrackingParameters)
{

	unsigned int nodeInd = blockIdx.x;
	unsigned int threadInd = threadIdx.x;

	const int nodeW = cameraTrackingParameters.nodeWidth;
	const int nodeH = cameraTrackingParameters.nodeHeight;
	const int nodeIndX = nodeInd % nodeW;
	const int nodeIndY = nodeInd / nodeW;
	const int imageW = cameraTrackingParameters.imageWidth;
	const int imageH = cameraTrackingParameters.imageHeight;
	const int localWindowHWidth = cameraTrackingParameters.localWindowHWidth;
	const float2 cellWH = cameraTrackingParameters.cellWH;

	float2 nodePos = cellWH * make_float2(nodeIndX, nodeIndY) + cameraTrackingParameters.offset;
	int2 nodePosi = make_int2(nodePos);

	int six = max(0, nodePosi.x - localWindowHWidth);
	int eix = min(imageW - 1, nodePosi.x + localWindowHWidth + 1);
	int siy = max(0, nodePosi.y - localWindowHWidth);
	int eiy = min(imageH - 1, nodePosi.y + localWindowHWidth + 1);

	int rangeW = eix - six; // sx <= x < ex
	int rangeH = eiy - siy; // sy <= y < ey
	int rangeN = rangeW * rangeH;

	float3 xRot = d_x_rot[nodeInd] + d_x_step_rot[nodeInd];
	float3 xTrans = d_x_trans[nodeInd] + d_x_step_trans[nodeInd];

	for (int i = threadInd; i < rangeN; i += BLOCK_SIZE) {

		int offix = i % rangeW;
		int offiy = i / rangeW;
		int ix = six + offix;
		int iy = siy + offiy;
		int index1D = iy * imageW + ix;

		mat3x1 pInput = mat3x1(make_float3(cameraTrackingInput.d_inputPos[index1D]));
		mat3x1 nInput = mat3x1(make_float3(cameraTrackingInput.d_inputNormal[index1D]));
		mat3x1 iInput = mat3x1(make_float3(cameraTrackingInput.d_inputIntensity4[index1D]));
		float maskVal = cameraTrackingInput.d_inputMask[index1D];

		//if (!pInput.checkMINF() && !nInput.checkMINF() && !iInput.checkMINF() && maskVal < 0.5 && iInput(0) + iInput(1) + iInput(2) >= 1.f / 255.f)
		if (!pInput.checkMINF() && !nInput.checkMINF() && !iInput.checkMINF() && maskVal < 0.5 && iInput.norm1DSquared() >= 0.8f)
		{
			float3 aRot, aTrans;

			mat3x3 I = mat3x3(intrinsics);
			mat3x3 ROld = mat3x3(evalRMat(xRot));
			mat3x1 pInputTransformed = ROld * pInput + mat3x1(xTrans);
			mat3x1 nInputTransformed = ROld * nInput;

			mat3x3 Ralpha = mat3x3(evalR_dGamma(xRot));
			mat3x3 Rbeta = mat3x3(evalR_dBeta(xRot));
			mat3x3 Rgamma = mat3x3(evalR_dAlpha(xRot));

			mat3x1 pProjTrans = I * pInputTransformed;

			if (pProjTrans(2) > 0.0f)
			{
				mat2x1 uvModel = mat2x1(dehomogenize(pProjTrans));

				mat3x1 iTargetAndDerivativeX = mat3x1(make_float3(bilinearInterpolationFloat4(uvModel(0), uvModel(1), cameraTrackingInput.d_targetIntensityAndDerivativesX, imageW, imageH)));
				mat3x1 iTargetAndDerivativeY = mat3x1(make_float3(bilinearInterpolationFloat4(uvModel(0), uvModel(1), cameraTrackingInput.d_targetIntensityAndDerivativesY, imageW, imageH)));
				mat3x1 iTargetAndDerivativeZ = mat3x1(make_float3(bilinearInterpolationFloat4(uvModel(0), uvModel(1), cameraTrackingInput.d_targetIntensityAndDerivativesZ, imageW, imageH)));

				mat3x1 iTarget = mat3x1(make_float3(iTargetAndDerivativeX(0), iTargetAndDerivativeY(0), iTargetAndDerivativeZ(0)));
				mat3x1 iTargetDerivUIntensity = mat3x1(make_float3(iTargetAndDerivativeX(1), iTargetAndDerivativeY(1), iTargetAndDerivativeZ(1)));
				mat3x1 iTargetDerivVIntensity = mat3x1(make_float3(iTargetAndDerivativeX(2), iTargetAndDerivativeY(2), iTargetAndDerivativeZ(2)));

				mat2x3 PI = dehomogenizeDerivative(pProjTrans);
				mat3x1 phiAlpha = Ralpha * pInputTransformed; mat3x1 phiBeta = Rbeta * pInputTransformed;	mat3x1 phiGamma = Rgamma * pInputTransformed;

				if (!iTarget.checkMINF() && !iTargetDerivUIntensity.checkMINF() && !iTargetDerivVIntensity.checkMINF())
				{
					// L2 loss
					{
						// Color
						mat3x1 diffIntensity(iTarget - iInput);
						mat3x2 DIntensity;
						DIntensity.setBlock(iTargetDerivUIntensity, 0, 0);
						DIntensity.setBlock(iTargetDerivVIntensity, 0, 1);

						mat1x2 DIntensityX; DIntensity.getBlock(0, 0, DIntensityX);
						mat1x2 DIntensityY; DIntensity.getBlock(1, 0, DIntensityY);
						mat1x2 DIntensityZ; DIntensity.getBlock(2, 0, DIntensityZ);
						if (DIntensityX.norm1D() + DIntensityY.norm1D() + DIntensityZ.norm1D() > cameraTrackingParameters.colorGradiantMin)
						{
							float weightColor;
							weightColor = weightDist(make_float2(ix, iy), nodePos, cameraTrackingParameters.sigma);
							//*max(0.0f, 1.0f - diffIntensity.norm1D() / cameraTrackingParameters.colorThres);;
							mat3x3 tmp0Intensity = DIntensity * PI*I;

							mat3x6 jacobianBlockRowIntensity;
							jacobianBlockRowIntensity.setBlock(tmp0Intensity*phiAlpha, 0, 0);
							jacobianBlockRowIntensity.setBlock(tmp0Intensity*phiBeta, 0, 1);
							jacobianBlockRowIntensity.setBlock(tmp0Intensity*phiGamma, 0, 2);
							jacobianBlockRowIntensity.setBlock(tmp0Intensity, 0, 3);
							addToLocalSystemGlobal3(jacobianBlockRowIntensity, diffIntensity, weightColor, nodeInd, threadIdx.x, d_temp);

						}
					}
				}
			}
		}
	}

	__syncthreads();


	// Up sweep 2D
#pragma unroll
	for (unsigned int stride = BLOCK_SIZE / 2; stride > 32; stride >>= 1)
	{
		if (threadIdx.x < stride) addToLocalScanElementGlobal(nodeInd, threadIdx.x + stride / 2, threadIdx.x, d_temp);

		__syncthreads();

	}
	if (threadIdx.x < 32) warpReduceGlobal(nodeInd, threadIdx.x, d_temp);


	//
	__syncthreads();

	// Copy to output texture
	if (threadIdx.x == 0) {

		CopyToResultScanElementGlobal(nodeInd, d_system, d_temp, lambda, d_x_step_rot[nodeInd], d_x_step_trans[nodeInd]);

	}
}

__global__ void scanNormal4EquationsLocalParallelRegDevice(unsigned int imageWidth, unsigned int imageHeight, float* d_system, float *d_temp, float3* d_x_rot, float3* d_x_trans, float3* d_x_step_rot, float3* d_x_step_trans, float lambda, CameraTrackingLocalInput cameraTrackingInput, float3x3 intrinsics, CameraTrackingLocalParameters cameraTrackingParameters)
{
	unsigned int nodeInd = blockIdx.x;
	unsigned int threadInd = threadIdx.x;

	const int nodeW = cameraTrackingParameters.nodeWidth;
	const int nodeH = cameraTrackingParameters.nodeHeight;
	const int nodeIndX = nodeInd % nodeW;
	const int nodeIndY = nodeInd / nodeW;
	const int imageW = cameraTrackingParameters.imageWidth;
	const int imageH = cameraTrackingParameters.imageHeight;
	const int localWindowHWidth = cameraTrackingParameters.localWindowHWidth;
	const float2 cellWH = cameraTrackingParameters.cellWH;

	float2 nodePos = cellWH * make_float2(nodeIndX, nodeIndY) + cameraTrackingParameters.offset;
	int2 nodePosi = make_int2(nodePos);

	int six = max(0, nodePosi.x - localWindowHWidth);
	int eix = min(imageW - 1, nodePosi.x + localWindowHWidth + 1);
	int siy = max(0, nodePosi.y - localWindowHWidth);
	int eiy = min(imageH - 1, nodePosi.y + localWindowHWidth + 1);

	int rangeW = eix - six; // sx <= x < ex
	int rangeH = eiy - siy; // sy <= y < ey
	int rangeN = rangeW * rangeH;

	float3 xRot = d_x_rot[nodeInd] + d_x_step_rot[nodeInd];
	float3 xTrans = d_x_trans[nodeInd] + d_x_step_trans[nodeInd];

	for (int i = threadInd; i < rangeN; i += BLOCK_SIZE) {

		int offix = i % rangeW;
		int offiy = i / rangeW;
		int ix = six + offix;
		int iy = siy + offiy;
		int index1D = iy * imageW + ix;

		mat3x1 pInput = mat3x1(make_float3(cameraTrackingInput.d_inputPos[index1D]));
		mat3x1 nInput = mat3x1(make_float3(cameraTrackingInput.d_inputNormal[index1D]));
		mat4x1 iInput = mat4x1(cameraTrackingInput.d_inputIntensity4[index1D]);
		float maskVal = cameraTrackingInput.d_inputMask[index1D];

		if (!pInput.checkMINF() && !nInput.checkMINF() && !iInput.checkMINF() && maskVal < 0.5 && iInput(3) >= 1.f / 255.f)
		{
			float3 aRot, aTrans;

			mat3x3 I = mat3x3(intrinsics);
			mat3x3 ROld = mat3x3(evalRMat(xRot));
			mat3x1 pInputTransformed = ROld * pInput + mat3x1(xTrans);
			mat3x1 nInputTransformed = ROld * nInput;

			mat3x3 Ralpha = mat3x3(evalR_dGamma(xRot));
			mat3x3 Rbeta = mat3x3(evalR_dBeta(xRot));
			mat3x3 Rgamma = mat3x3(evalR_dAlpha(xRot));

			mat3x1 pProjTrans = I * pInputTransformed;

			if (pProjTrans(2) > 0.0f)
			{
				mat2x1 uvModel = mat2x1(dehomogenize(pProjTrans));

				mat3x1 iTargetAndDerivativeX = mat3x1(make_float3(bilinearInterpolationFloat4(uvModel(0), uvModel(1), cameraTrackingInput.d_targetIntensityAndDerivativesX, imageW, imageH)));
				mat3x1 iTargetAndDerivativeY = mat3x1(make_float3(bilinearInterpolationFloat4(uvModel(0), uvModel(1), cameraTrackingInput.d_targetIntensityAndDerivativesY, imageW, imageH)));
				mat3x1 iTargetAndDerivativeZ = mat3x1(make_float3(bilinearInterpolationFloat4(uvModel(0), uvModel(1), cameraTrackingInput.d_targetIntensityAndDerivativesZ, imageW, imageH)));
				mat3x1 iTargetAndDerivativeW = mat3x1(make_float3(bilinearInterpolationFloat4(uvModel(0), uvModel(1), cameraTrackingInput.d_targetIntensityAndDerivativesW, imageW, imageH)));

				mat4x1 iTarget = mat4x1(make_float4(iTargetAndDerivativeX(0), iTargetAndDerivativeY(0), iTargetAndDerivativeZ(0), iTargetAndDerivativeW(0)));
				mat4x1 iTargetDerivUIntensity = mat4x1(make_float4(iTargetAndDerivativeX(1), iTargetAndDerivativeY(1), iTargetAndDerivativeZ(1), iTargetAndDerivativeW(1)));
				mat4x1 iTargetDerivVIntensity = mat4x1(make_float4(iTargetAndDerivativeX(2), iTargetAndDerivativeY(2), iTargetAndDerivativeZ(2), iTargetAndDerivativeW(2)));

				mat2x3 PI = dehomogenizeDerivative(pProjTrans);
				mat3x1 phiAlpha = Ralpha * pInputTransformed; mat3x1 phiBeta = Rbeta * pInputTransformed;	mat3x1 phiGamma = Rgamma * pInputTransformed;

				if (!iTarget.checkMINF() && !iTargetDerivUIntensity.checkMINF() && !iTargetDerivVIntensity.checkMINF())
				{
					// Color
					mat4x1 diffIntensity(iTarget - iInput);
					mat3x1 diffNormal;
					diffIntensity.getBlock(0, 0, diffNormal);

					matNxM<4, 2> DIntensity;
					DIntensity.setBlock(iTargetDerivUIntensity, 0, 0);
					DIntensity.setBlock(iTargetDerivVIntensity, 0, 1);

					mat1x2 DIntensityX; DIntensity.getBlock(0, 0, DIntensityX);
					mat1x2 DIntensityY; DIntensity.getBlock(1, 0, DIntensityY);
					mat1x2 DIntensityZ; DIntensity.getBlock(2, 0, DIntensityZ);
					mat1x2 DIntensityW; DIntensity.getBlock(3, 0, DIntensityW);
					if (DIntensityX.norm1D() + DIntensityY.norm1D() + DIntensityZ.norm1D() + DIntensityW.norm1D() > cameraTrackingParameters.colorGradiantMin)
					{
						{
							float weightNormal;
							weightNormal = weightDist(make_float2(ix, iy), nodePos, cameraTrackingParameters.sigma) * cameraTrackingParameters.lambdaNormals;
							mat3x2 DIntensity3;
							DIntensity.getBlock(0, 0, DIntensity3);
							mat3x3 tmp0Intensity = DIntensity3 * PI*I;

							mat3x6 jacobianBlockRowIntensity;
							jacobianBlockRowIntensity.setBlock(tmp0Intensity*phiAlpha, 0, 0);
							jacobianBlockRowIntensity.setBlock(tmp0Intensity*phiBeta, 0, 1);
							jacobianBlockRowIntensity.setBlock(tmp0Intensity*phiGamma, 0, 2);
							jacobianBlockRowIntensity.setBlock(tmp0Intensity, 0, 3);
							addToLocalSystemGlobal3(jacobianBlockRowIntensity, diffNormal, weightNormal, nodeInd, threadIdx.x, d_temp);
						}
						{
							float weightColor = weightDist(make_float2(ix, iy), nodePos, cameraTrackingParameters.sigma) * cameraTrackingParameters.lambdaColors;

							mat1x3 tmp0Intensity = DIntensityW * PI*I;

							mat1x6 jacobianBlockRowIntensity;
							jacobianBlockRowIntensity.setBlock(tmp0Intensity*phiAlpha, 0, 0);
							jacobianBlockRowIntensity.setBlock(tmp0Intensity*phiBeta, 0, 1);
							jacobianBlockRowIntensity.setBlock(tmp0Intensity*phiGamma, 0, 2);
							jacobianBlockRowIntensity.setBlock(tmp0Intensity, 0, 3);
							addToLocalSystemGlobal(jacobianBlockRowIntensity, mat1x1(diffIntensity(3)), weightColor, nodeInd, threadIdx.x, d_temp);
						}
					}
				}
			}
		}
	}

	__syncthreads();


	// Up sweep 2D
#pragma unroll
	for (unsigned int stride = BLOCK_SIZE / 2; stride > 32; stride >>= 1)
	{
		if (threadIdx.x < stride) addToLocalScanElementGlobal(nodeInd, threadIdx.x + stride / 2, threadIdx.x, d_temp);

		__syncthreads();

	}
	if (threadIdx.x < 32) warpReduceGlobal(nodeInd, threadIdx.x, d_temp);


	//
	__syncthreads();

	// Copy to output texture
	if (threadIdx.x == 0) {

		CopyToResultScanElementGlobal(nodeInd, d_system, d_temp, lambda, d_x_step_rot[nodeInd], d_x_step_trans[nodeInd]);

	}
}

extern "C" void computeNormalEquationsLocal(unsigned int imageWidth, unsigned int imageHeight, float* output, CameraTrackingLocalInput cameraTrackingInput, float* intrinsics, CameraTrackingLocalParameters cameraTrackingParameters, float3 anglesOld, float3 translationOld, float2 pos, unsigned int localWindowSize, unsigned int blockSizeInt)
{
	const int localWindowWidth = cameraTrackingParameters.localWindowHWidth * 2 + 1;
	const unsigned int numElements = localWindowWidth * localWindowWidth;
	dim3 blockSize(blockSizeInt, 1, 1);
	dim3 gridSize((numElements + blockSizeInt * localWindowSize - 1) / (blockSizeInt*localWindowSize), 1, 1);

	scanNormalEquationsLocalDevice << <gridSize, blockSize >> > (imageWidth, imageHeight, output, cameraTrackingInput, float3x3(intrinsics), cameraTrackingParameters, anglesOld, translationOld, pos, localWindowSize);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}


extern "C" void computeNormalEquationsAllLocal(unsigned int imageWidth, unsigned int imageHeight, float* d_system, float *d_temp, float3 *d_x_rot, float3 *d_x_trans, CameraTrackingLocalInput cameraTrackingInput, float* intrinsics, CameraTrackingLocalParameters cameraTrackingParameters) {


	const int nodeN = cameraTrackingParameters.nodeHeight * cameraTrackingParameters.nodeWidth;


	dim3 blockSize(BLOCK_SIZE, 1, 1);
	dim3 gridSize(nodeN, 1, 1);

	hipMemset(d_temp, 0, sizeof(float)* BLOCK_SIZE * ARRAY_SIZE * nodeN);

	scanNormalEquationsLocalParallelDevice << <gridSize, blockSize >> > (imageWidth, imageHeight, d_system, d_temp, d_x_rot, d_x_trans, cameraTrackingInput, float3x3(intrinsics), cameraTrackingParameters);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

extern "C" void computeNormal3EquationsAllRegLocal(unsigned int imageWidth, unsigned int imageHeight, float* d_system, float *d_temp, float3 *d_x_rot, float3 *d_x_trans, float3 *d_x_step_rot, float3 *d_x_step_trans, float lambda, CameraTrackingLocalInput cameraTrackingInput, float* intrinsics, CameraTrackingLocalParameters cameraTrackingParameters) {


	const int nodeN = cameraTrackingParameters.nodeHeight * cameraTrackingParameters.nodeWidth;


	dim3 blockSize(BLOCK_SIZE, 1, 1);
	dim3 gridSize(nodeN, 1, 1);

	hipMemset(d_temp, 0, sizeof(float)* BLOCK_SIZE * ARRAY_SIZE * nodeN);

	scanNormal3EquationsLocalParallelRegDevice << <gridSize, blockSize >> > (imageWidth, imageHeight, d_system, d_temp, d_x_rot, d_x_trans, d_x_step_rot, d_x_step_trans, lambda, cameraTrackingInput, float3x3(intrinsics), cameraTrackingParameters);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

extern "C" void computeNormal4EquationsAllRegLocal(unsigned int imageWidth, unsigned int imageHeight, float* d_system, float *d_temp, float3 *d_x_rot, float3 *d_x_trans, float3 *d_x_step_rot, float3 *d_x_step_trans, float lambda, CameraTrackingLocalInput cameraTrackingInput, float* intrinsics, CameraTrackingLocalParameters cameraTrackingParameters) {


	const int nodeN = cameraTrackingParameters.nodeHeight * cameraTrackingParameters.nodeWidth;


	dim3 blockSize(BLOCK_SIZE, 1, 1);
	dim3 gridSize(nodeN, 1, 1);

	hipMemset(d_temp, 0, sizeof(float)* BLOCK_SIZE * ARRAY_SIZE * nodeN);

	scanNormal4EquationsLocalParallelRegDevice << <gridSize, blockSize >> > (imageWidth, imageHeight, d_system, d_temp, d_x_rot, d_x_trans, d_x_step_rot, d_x_step_trans, lambda, cameraTrackingInput, float3x3(intrinsics), cameraTrackingParameters);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

extern "C" void computeNormalEquationsAllRegLocal(unsigned int imageWidth, unsigned int imageHeight, float* d_system, float *d_temp, float3 *d_x_rot, float3 *d_x_trans, float3 *d_x_step_rot, float3 *d_x_step_trans, float lambda, CameraTrackingLocalInput cameraTrackingInput, float* intrinsics, CameraTrackingLocalParameters cameraTrackingParameters) {


	const int nodeN = cameraTrackingParameters.nodeHeight * cameraTrackingParameters.nodeWidth;


	dim3 blockSize(BLOCK_SIZE, 1, 1);
	dim3 gridSize(nodeN, 1, 1);

	hipMemset(d_temp, 0, sizeof(float)* BLOCK_SIZE * ARRAY_SIZE * nodeN);

	scanNormalEquationsLocalParallelRegDevice << <gridSize, blockSize >> > (imageWidth, imageHeight, d_system, d_temp, d_x_rot, d_x_trans, d_x_step_rot, d_x_step_trans, lambda, cameraTrackingInput, float3x3(intrinsics), cameraTrackingParameters);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}