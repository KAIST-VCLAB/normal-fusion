
#include "CUDAHashParams.h"
#include "CUDARayCastParams.h"
#include "CUDADepthCameraParams.h"
#include "CUDATexPoolParams.h"
#include "CUDATexUpdateParams.h"
#include "CUDAInputEnhancementParams.h"

__constant__ HashParams c_hashParams;
__constant__ RayCastParams c_rayCastParams;
__constant__ DepthCameraParams c_depthCameraParams;
__constant__ TexPoolParams c_texPoolParams;
__constant__ TexUpdateParams c_texUpdateParams;
__constant__ InputEnhanceParams c_inputEnhanceParams;


extern "C" void updateConstantInputEnhanceParams(const InputEnhanceParams& params) {

	size_t size;
	cutilSafeCall(hipGetSymbolSize(&size, HIP_SYMBOL(c_inputEnhanceParams)));
	cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_inputEnhanceParams), &params, size, 0, hipMemcpyHostToDevice));

#ifdef DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

extern "C" void updateConstantTexUpdateParams(const TexUpdateParams& params) {

	size_t size;
	cutilSafeCall(hipGetSymbolSize(&size, HIP_SYMBOL(c_texUpdateParams)));
	cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_texUpdateParams), &params, size, 0, hipMemcpyHostToDevice));

#ifdef DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

extern "C" void updateConstantTexPoolParams(const TexPoolParams& params) {

	size_t size;
	cutilSafeCall(hipGetSymbolSize(&size, HIP_SYMBOL(c_texPoolParams)));
	cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_texPoolParams), &params, size, 0, hipMemcpyHostToDevice));

#ifdef DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

extern "C" void updateConstantHashParams(const HashParams& params) {

	size_t size;
	cutilSafeCall(hipGetSymbolSize(&size, HIP_SYMBOL(c_hashParams)));
	cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_hashParams), &params, size, 0, hipMemcpyHostToDevice));
	
#ifdef DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
	}


extern "C" void updateConstantRayCastParams(const RayCastParams& params) {
	
	size_t size;
	cutilSafeCall(hipGetSymbolSize(&size, HIP_SYMBOL(c_rayCastParams)));
	cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_rayCastParams), &params, size, 0, hipMemcpyHostToDevice));
	
#ifdef DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif

}

extern "C" void updateConstantDepthCameraParams(const DepthCameraParams& params) {
	
	size_t size;
	cutilSafeCall(hipGetSymbolSize(&size, HIP_SYMBOL(c_depthCameraParams)));
	cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_depthCameraParams), &params, size, 0, hipMemcpyHostToDevice));
	
#ifdef DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif

}

